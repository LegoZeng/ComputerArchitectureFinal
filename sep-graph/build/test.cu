#include<hip/hip_runtime.h>
#include<nvml.h>
#include<stdio.h>
int main() {
    // 初始化NvML库
    nvmlReturn_t result = nvmlInit();
    if (result != NVML_SUCCESS) {
        // 错误处理
        return 1;
    }

    // 获取GPU数量
    unsigned int deviceCount;
    result = nvmlDeviceGetCount(&deviceCount);
    if (result != NVML_SUCCESS) {
        // 错误处理
        nvmlShutdown();
        return 1;
    }

    // 获取每个GPU的利用率等信息
    for (unsigned int i = 0; i < deviceCount; ++i) {
        nvmlDevice_t device;
        result = nvmlDeviceGetHandleByIndex(i, &device);
        if (result != NVML_SUCCESS) {
            // 错误处理
            nvmlShutdown();
            return 1;
        }

        nvmlUtilization_t utilization;
        result = nvmlDeviceGetUtilizationRates(device, &utilization);
        if (result != NVML_SUCCESS) {
            // 错误处理
            nvmlShutdown();
            return 1;
        }

        // 输出GPU的利用率等信息
        printf("GPU %d Utilization: %d%%\n", i, utilization.gpu);

        printf("GPU memory %d Utilization: %d%%\n", i, utilization.memory);
    }

    // 关闭NvML库
    nvmlShutdown();

    return 0;
}





